#include "hip/hip_runtime.h"
#include "vinegar.h"

__global__
void vecAddKernel(float * A, float *B, float *C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n)
    C[i] = A[i] + B[i];
}

void vecAdd(float *A, float *B, float *C, int n) {
  float *A_d, *B_d, *C_d;
  size_t size = sizeof(float[n]);

  CHECKED_CUDA_API(hipMalloc((void **) &A_d,size));
  CHECKED_CUDA_API(hipMalloc((void **) &B_d,size));
  CHECKED_CUDA_API(hipMalloc((void **) &C_d,size));

  CHECKED_CUDA_API(hipMemcpy(A_d,A,size,hipMemcpyHostToDevice));
  CHECKED_CUDA_API(hipMemcpy(B_d,B,size,hipMemcpyHostToDevice));

  vecAddKernel <<<ceil(n/256.0), 256>>>(A_d,B_d,C_d,n);

  CHECKED_CUDA_API(hipMemcpy(C,C_d,size,hipMemcpyDeviceToHost));

  CHECKED_CUDA_API(hipFree(A_d));
  CHECKED_CUDA_API(hipFree(B_d));
  CHECKED_CUDA_API(hipFree(C_d));

}

int main() {
  float A[100],B[100],C[100];
  int i=0;
  for(i=0;i<100;i++) {
    A[i] = (float)i+1;
    B[i] = (float)(100-i);
  }

  vecAdd(A,B,C,100);

  for(i=0;i<100;i++) {
    if(fabs(C[i] - 100.0) > 0.01) {
      printf("Addition failed for : %d \n",i);
      exit(255);
    }
  }

  printf("Addition success ! \n");
}

