#include "hip/hip_runtime.h"
#include "vinegar.h"

__global__
void vecAddKernel(float * A, float *B, float *C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n)
    C[i] = A[i] + B[i];
}

void vecAdd(float *A, float *B, float *C, int n) {
  float *A_d, *B_d, *C_d;
  size_t size = sizeof(float[n]);

  CHECKED_CUDA_API(hipMalloc((void **) &A_d,size));
  CHECKED_CUDA_API(hipMalloc((void **) &B_d,size));
  CHECKED_CUDA_API(hipMalloc((void **) &C_d,size));

  CHECKED_CUDA_API(hipMemcpy(A_d,A,size,hipMemcpyHostToDevice));
  CHECKED_CUDA_API(hipMemcpy(B_d,B,size,hipMemcpyHostToDevice));

  vecAddKernel <<<ceil(n/256.0), 256>>>(A_d,B_d,C_d,n);

  CHECKED_CUDA_API(hipMemcpy(C,C_d,size,hipMemcpyDeviceToHost));

  CHECKED_CUDA_API(hipFree(A_d));
  CHECKED_CUDA_API(hipFree(B_d));
  CHECKED_CUDA_API(hipFree(C_d));

}

